#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

// Use Thrust to handle host/device allocations
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

// Cutlass includes
#include <cutlass/half.h>                       // F16 data type
#include <cutlass/util/print_error.hpp>
#include <cutlass/arch/barrier.h>
#include <cutlass/cluster_launch.hpp>

// CuTe includes
#include <cute/tensor.hpp>                      // CuTe tensor implementation
#include <cute/numeric/integral_constant.hpp>   // Compile time in constants such as _1, _256 etc.
#include <cute/algorithm/cooperative_copy.hpp>  // Auto vectorized copy operation

using namespace cute;

int main() {
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    int computeCapability = prop.major * 10 + prop.minor;
    std::cout << "Compute capability: " << computeCapability << std::endl;

    if (computeCapability == 120) {
        std::cout << "Nice." << std::endl;
        return 0;
    } else {
        std::cout << "The program requires a GPU with compute capability of 12.0" << std::endl;
        return -1;
    }
}
